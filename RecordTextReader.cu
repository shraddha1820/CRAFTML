
#include <hip/hip_runtime.h>
#include<bits/stdc++.h>
#include <iostream>
#include <fstream>

using namespace std;

class RecordTextReader
{
    public:

    long nbReadLine=0;


	ifstream reader;

	string separatorInText = " ";
	string separatorRecord =" ";
	
    bool fistColumnIsID=false;
    bool cleanLineOption=true;
    bool lowerCase=true;
    bool upperCase=false;
    bool recordMode=true;
    bool tryMode=false;
	
	bool secondFieldIsText=true;
	
	
	bool deleteAfterSlash=true;
	

			
	// public string cleanText(string myText) {

	// 	 {
	// 		//line=line.trim();
	// 		if (lowerCase) {
	// 			myText=myText.toLowerCase();   
	// 		}
	// 		if (upperCase) {
	// 			myText=myText.toUpperCase();   
				
	// 		}
	// 		myText=myText.replaceAll("\t", " ");
	// 		myText=myText.replace("(", " ");
	// 		myText=myText.replace(")", " ");
	// 		myText=myText.replace(",", " ");
	// 		myText=myText.replace(";", " ");
	// 		myText=myText.replace(".", " ");    //TODO : v�rifier l'expresseion r�guli�re pour le point
	// 		myText=myText.replaceAll("    ", " ");
	// 		myText=myText.replaceAll("   ", " ");
	// 		myText=myText.replaceAll("  ", " ");
	// 		//line=line.replace(commandTag2, commandTag1);
			
	// 		if (deleteAfterSlash) {
	// 			int p;
	// 			p=myText.indexOf("/");
	// 			if (p>0) {  // pas retir� si en d�but de chaine, au cas o�
	// 				//System.out.println("TEXT:"+myText);
	// 				myText=myText.substring(0, p);
	// 			}
	// 			//System.out.println("TEXT:"+myText);
	// 			//System.exit(1);
	// 		}
			
			
	// 		myText=myText.trim();
	// 		return myText;
	// 	}
	// }
			
			

	// public string getSeparatorRecord() {
	// 	return separatorRecord;
	// }



	// public void setSeparatorRecord(string separatorRecord) {
	// 	this.separatorRecord = separatorRecord;
	// }



	bool openFile(string filename)
	{
		
		cout<<"\n Opening file: "<<filename<<"\n";
		
		
		/*
		System.out.println("ID in 1st column:"+fistColumnIsID);
		if (fistColumnIsID) {
			System.out.println("record separator:["+separatorRecord+"]");
		}
		System.out.println("with text separator:["+separatorInText+"]");
		System.out.println("cleaning :"+cleanLineOption);
		if (cleanLineOption) {
			if (lowerCase) {
			  System.out.println("     to lowercase");
			}
			if (upperCase) {
				System.out.println("    to upperCase");
			}
		}
		
		*/
		
		try
		{
			this->reader.open(filename);
			this->nbReadLine=0;
			return true;
		}
		catch(int x)
		{
			cout<<filename<<" can't be opened"<<endl;
		}
		return false;
	}




	// public void closeFile()
	// {
	// 	if(reader != null) {
	// 		try {
	// 			reader.close();
	// 		} catch (IOException e) {
	// 			e.printStackTrace();
	// 		}
	// 	}
	// }
	
	
	// public string readLine(){
	// 	try
	// 	{
	// 		string line;
	// 		//System.out.println("va lire...");
	// 		if ((line = reader.readLine()) != null)
	// 		{
	// 			return line;
	// 		}
	// 	}
	// 	catch (Exception e)
	// 	{
	// 			e.printStackTrace();
	// 	}
	// 	return null;
	// }
	vector<string> split(string s,string del)
	{
		stringstream ss(s);
		vector<string> str;
		string temp;
		while (getline(ss, temp, del[0])) { 
			str.push_back(temp); 
		} 
		return str;
	}

	vector<string> readPureRecord()
	{
		try
		{
			string line;
			//System.out.println("va lire...");
			if ( getline (this->reader,line))
			{
				
				this->nbReadLine++;
				// if (tryMode) {
				// 	if (nbReadLine>5) {
				// 		Displayer.displayText("TRY MODE: exit");
				// 		reader.close();
				// 		return null;
				// 	}
				// }
				
				//Displayer.displayText("ligne lue:["+line+"]");
				
				// string token = s.substr(0, s.find(delimiter));
				vector<string> record=split(line,separatorRecord);
				// for(auto it: record)
				// {
				// 	cout<<it<<" ";
				// }
				// cout<<endl;
				/*
				 Iterable<string> cutLine = Splitter.on(separatorRecord).split(line);				
				string[] record = Iterables.toArray(cutLine, string.class);
				 */
				
				return record;
			}
			this->reader.close();    
		}
		catch (int e)
		{
				// e.printStackTrace();
		}
        vector<string> record(2,"");
		return record;
	}

	
	// public string[] readLineOrRecord()
	// {
	// 	try
	// 	{
	// 		string line;
	// 		//System.out.println("va lire...");
	// 		if ((line = reader.readLine()) != null)
	// 		{
				
	// 			nbReadLine++;
	// 			if (tryMode) {
	// 				if (nbReadLine>5) {
	// 					Displayer.displayText("TRY MODE: exit");
	// 					reader.close();
	// 					return null;
	// 				}
	// 			}
				
	// 			//Displayer.displayText("ligne lue:["+line+"]");
				
				
	// 			string[] record;
	// 			if (recordMode) {
	// 				if (!fistColumnIsID) {
	// 					record= line.split(separatorInText);
	// 				} else {
	// 					string[] idAndText=line.split(separatorRecord);
	// 					//if (cleanLineOption) {
	// 					//	idAndText[1]=cleanLine(idAndText[1]);
	// 					//	System.out.println("ligne 'clean�e' :["+idAndText[1]+"]");
	// 					//}
	// 					string[] textRecord;
	// 					if (idAndText.length>1) {
	// 						 textRecord=idAndText[1].split(separatorInText);
	// 					} else {
	// 						textRecord=new string[1];
	// 						textRecord[0]="";
	// 					}
	// 					if (cleanLineOption) {
	// 						for (int i=0;i<textRecord.length;i++) {
	// 							textRecord[i]=cleanText(textRecord[i]);
	// 						}
	// 					}
	// 					record=new string[textRecord.length+1];
	// 					record[0]=idAndText[0];
	// 					for (int i=0;i<textRecord.length;i++) {
	// 						record[i+1]=textRecord[i];
	// 					}
	// 				}
					
	// 			} else {
	// 				record=new string[1];
	// 				if (cleanLineOption) {
	// 					line=cleanText(line);
	// 					//Displayer.displayText("ligne 'clean�e' :["+line+"]");
	// 				}
	// 				record[0]=line;
	// 			}
	// 			return record;
	// 		}
	// 		reader.close();    
	// 	}
	// 	catch (Exception e)
	// 	{
	// 			e.printStackTrace();
	// 	}
	// 	return null;
	// }
	
	// static public void printRecord(string[] record,string separatorOut) {
	// 	if (record!=null) {
	// 		for (int i=0; i<record.length;i++) {
	// 			//Displayer.displayText(record[i]);
	// 			//Displayer.displayText(separatorOut);
	// 		}
	// 	}
	// 	//Displayer.displayText("");
	// }
    
};