#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include<string.h>
// #include "RecordTextReader.cpp"
#include "LibsvmFileReader.cu"
#include<math.h>
#include <iostream>
#include <fstream>
#include<hip/hip_runtime.h>
// #include "cuPrintf.cu"
// #include "cuPrintf.cuh"

#include <thrust/device_vector.h>
#include <thrust/copy.h>

using namespace std;

uint32_t MurmurHash2 ( const void * key, int len, uint32_t seed );
class CraftML
{
	public:
		
	vector<vector<float>> valuesX;
	vector<vector<int>> keyX;
	vector<vector<float>> valuesY;
	vector<vector<int>> keyY;
	
	void storeForLearning(unordered_map<int,float> x,unordered_map<int,float> y)
	{
		vector<float> valueX;
		vector<float> valueY;

		vector<int> indexX;
		vector<int> indexY;


		for (auto row : x) 
      	{
			indexX.push_back(row.first);
			valueX.push_back(row.second); 
		}

		for (auto row : y) 
      	{
			indexY.push_back(row.first);
			valueY.push_back(row.second); 
		}

		if(indexX.size()>0 && indexY.size()>0)
		{
			keyX.push_back(indexX);
			keyY.push_back(indexY);

			valuesX.push_back(valueX);
			valuesY.push_back(valueY);

		} 

	}
    void trainAlgoOnFileStandard(LibsvmFileReader &fileReader)
    {
        fileReader.openFile();
		
		// reinitialize();
		// SmallItem x;
		// SmallItem y;
		unordered_map<int,float> x;
		unordered_map<int,float> y;

		pair<unordered_map<int,float>,unordered_map<int,float>> x_y = fileReader.readNext();
		// boolean continueRead = fileReader.readNext();
		
		while(x_y.first.find(-1)==x_y.first.end())
		{
		// 	if(interrompre()) {
		// 		return;
		// 	}
			x = x_y.first;
			y = x_y.second;

			storeForLearning(x, y);
			x_y = fileReader.readNext();
		}
		// cout<<"No. of rows "<<keyY.size()<<endl;
		
		// for(int i=0;i<5;i++)
		// {
		// 	for(int j=0;j<keyY[i].size();j++)
		// 	{
		// 		cout<<keyY[i][j]<<":"<<valuesY[i][j]<<" ";
		// 	}
		// 	cout<<endl;
		// }
			

		// indicationFinDePasse();
		// hasLearnt = true;
		// fileReader.closeFile();
    }
    

};
struct Args_device
{
	float** valuesX;
	float** valuesY;
	int** indexX;
	int** indexY;
	int* useInst;
	int dimProjX;
	int dimProjY;
	int sparsity;
	int seedX;
	int seedY;
	// int len;
	// int *lenX;
	// int *lenY;
};

__device__ int get_length(int key)
{
	int len = 1;
	while(key>9)
	{
		len++;
		key/=10;
	}
	return len;
}

__device__ uint32_t MurmurHash2 ( const void * key, int len, uint32_t seed )
{
  /* 'm' and 'r' are mixing constants generated offline.
     They're not really 'magic', they just happen to work well.  */

  const uint32_t m = 0x5bd1e995;
  const int r = 24;

  /* Initialize the hash to a 'random' value */

  uint32_t h = seed ^ len;

  /* Mix 4 bytes at a time into the hash */

  const unsigned char * data = (const unsigned char *)key;

  while(len >= 4)
  {
    uint32_t k = *(uint32_t*)data;

    k *= m;
    k ^= k >> r;
    k *= m;

    h *= m;
    h ^= k;

    data += 4;
    len -= 4;
  }

  /* Handle the last few bytes of the input array  */

  switch(len)
  {
  case 3: h ^= data[2] << 16;
  case 2: h ^= data[1] << 8;
  case 1: h ^= data[0];
      h *= m;
  };

  /* Do a few final mixes of the hash to ensure the last few
  // bytes are well-incorporated.  */

  h ^= h >> 13;
  h *= m;
  h ^= h >> 15;

  return h;
} 

__device__ int getIndex(int key,int seed,int sizeMax){
	int len = get_length(key)+3;

	char* hashkey=new char[len];

	//key="azv"

	hashkey[0]='a';
	hashkey[1]='z';
	hashkey[2]='v';

	int i=1;
	while(key>0)
	{
		hashkey[len-i]=(char)(key%10+48);
		i++;
		key/=10;
	}

	return abs((int)MurmurHash2(hashkey,len,seed) % sizeMax);
}
__device__ float getSign(int key,int seed){
	const int len = get_length(key)+3;

	char* hashkey=new char[len];

	//key="azv"

	hashkey[0]='a';
	hashkey[1]='z';
	hashkey[2]='v';

	int i=1;
	while(key>0)
	{
		hashkey[len-i]=(char)(key%10+48);
		i++;
		key/=10;
	}


	return (float)abs((int)MurmurHash2(hashkey, len, seed) % 2)*2 - 1;
}
struct Rand_proj
{
	int *a[1000];
	// int *b[1000];
};
__global__ void buildTree(struct Args_device *d_args,struct Rand_proj *out)
{
	// *var=d_args->lenX[1000];
	for(int j = 0; j<d_args->useInst[0];j++){
		xKey = keyX.get(useInst.get(j));
		xValues = valuesX.get(useInst.get(j));

		if(xKey!=null) {

			projectedX = new float[dimXProj];

			for(int i = 0;i<xKey.length;i++){
				currentIndex = getIndex(xKey[i], seedIndexX, dimXProj);
				currentSign = getSign(xKey[i], seedSignX);
				projectedX[currentIndex] = projectedX[currentIndex] + currentSign * xValues[i];
			}

			float norm = 0;
			int nonZeros = 0;

			for(int i = 0;i<projectedX.length;i++){
				if(projectedX[i] != 0f) {
					norm += projectedX[i]*projectedX[i];
					nonZeros++;
				}
			}
		}
	}

}
int main(int argc,char** argv)
{
	string trainFile=argv[1];

    CraftML model ;

    LibsvmFileReader readerTrain;
					
	readerTrain.setFile(trainFile);

	model.trainAlgoOnFileStandard(readerTrain);

	struct Args_device *args;
	struct Args_device *d_args;
	struct Args_device *id1_args;
	struct Args_device *id2_args;

	args = (struct Args_device *)malloc(sizeof(struct Args_device));
	id1_args = (struct Args_device *)malloc(sizeof(struct Args_device));
	id2_args = (struct Args_device *)malloc(sizeof(struct Args_device));
	hipMalloc((void **)&d_args,sizeof(struct Args_device));


	args->indexX = (int **)malloc(model.keyX.size()*sizeof(int *));
	hipMalloc((void **)&(id1_args->indexX),model.keyX.size()*sizeof(int *));
	id2_args->indexX = (int **)malloc(model.keyX.size()*sizeof(int *));

	
	args->valuesX = (float **)malloc(model.keyX.size()*sizeof(float *));
	hipMalloc((void **)&(id1_args->valuesX),model.keyX.size()*sizeof(float *));
	id2_args->valuesX = (float **)malloc(model.keyX.size()*sizeof(float *));
	

	for(int i=0;i<model.keyX.size();i++)
	{
		args->indexX[i]=(int *)malloc((model.keyX[i].size()+1)*sizeof(int));
		hipMalloc((void **)&(id2_args->indexX[i]),(model.keyX[i].size()+1)*sizeof(int));

		args->valuesX[i]=(float *)malloc((model.keyX[i].size()+1)*sizeof(float));
		hipMalloc((void **)&(id2_args->valuesX[i]),(model.keyX[i].size()+1)*sizeof(float));

		args->indexX[i][0]=model.keyX[i].size();
		args->valuesX[i][0]=model.keyX[i].size();

		for(int j=0;j<model.keyX[i].size();j++)
		{
			args->indexX[i][j+1] = model.keyX[i][j];
			args->valuesX[i][j+1] = model.valuesX[i][j];
		}
		hipMemcpy(id2_args->indexX[i],args->indexX[i],model.keyX[i].size()+1,hipMemcpyHostToDevice);
		hipMemcpy(id2_args->valuesX[i],args->valuesX[i],model.keyX[i].size()+1,hipMemcpyHostToDevice);
	}
	hipMemcpy(id1_args->indexX,id2_args->indexX,model.keyX.size(),hipMemcpyHostToDevice);
	hipMemcpy(id1_args->valuesX,id2_args->valuesX,model.keyX.size(),hipMemcpyHostToDevice);

	args->indexY = (int **)malloc(model.keyY.size()*sizeof(int *));
	hipMalloc((void **)&(id1_args->indexY),model.keyY.size()*sizeof(int *));
	id2_args->indexY = (int **)malloc(model.keyY.size()*sizeof(int *));

	args->valuesY = (float **)malloc(model.keyY.size()*sizeof(float *));
	hipMalloc((void **)&(id1_args->valuesY),model.keyY.size()*sizeof(float *));
	id2_args->valuesY = (float **)malloc(model.keyY.size()*sizeof(float *));

	for(int i=0;i<model.keyY.size();i++)
	{
		args->indexY[i]=(int *)malloc((model.keyY[i].size()+1)*sizeof(int));
		hipMalloc((void **)&(id2_args->indexY[i]),(model.keyY[i].size()+1)*sizeof(int));

		args->valuesY[i]=(float *)malloc((model.keyY[i].size()+1)*sizeof(float));
		hipMalloc((void **)&(id2_args->valuesY[i]),(model.keyY[i].size()+1)*sizeof(float));

		args->indexY[i][0]=model.keyY[i].size();
		args->valuesY[i][0]=model.keyY[i].size();

		for(int j=0;j<model.keyY[i].size();j++)
		{
			args->indexY[i][j+1] = model.keyY[i][j];
			args->valuesY[i][j+1] = model.valuesY[i][j];
		}
		hipMemcpy(id2_args->indexY[i],args->indexY[i],model.keyY[i].size()+1,hipMemcpyHostToDevice);
		hipMemcpy(id2_args->valuesY[i],args->valuesY[i],model.keyY[i].size()+1,hipMemcpyHostToDevice);

	}
	hipMemcpy(id1_args->indexY,id2_args->indexY,model.keyY.size(),hipMemcpyHostToDevice);
	hipMemcpy(id1_args->valuesY,id2_args->valuesY,model.keyY.size(),hipMemcpyHostToDevice);

	args->useInst = (int *)malloc((model.keyY.size()+1)*sizeof(int));
	args->useInst[0]=keyX.size();
	for(int i=0;i<model.keyX.size();i++)
		args->useInst[i+1] = i;
	hipMalloc((void **)&(id1_args->useInst),(model.keyY.size()+1)*sizeof(int));
	hipMemcpy(id1_args->useInst,args->useInst,model.keyX.size()+1,hipMemcpyHostToDevice);
	
	// args->lenX = (int *)malloc(model.keyY.size()*sizeof(int));
	// for(int i=0;i<model.keyX.size();i++)
	// 	args->lenX[i] = model.keyX[i].size();
	// hipMalloc((void **)&(id1_args->lenX),model.keyY.size()*sizeof(int));
	// hipMemcpy(id1_args->lenX,args->lenX,model.keyX.size(),hipMemcpyHostToDevice);
	
	// args->lenY = (int *)malloc(model.keyY.size()*sizeof(int));
	// for(int i=0;i<model.keyX.size();i++)
	// 	args->lenY[i] = model.keyY[i].size();
	// hipMalloc((void **)&(id1_args->lenY),model.keyY.size()*sizeof(int));
	// hipMemcpy(id1_args->lenY,args->lenY,model.keyX.size(),hipMemcpyHostToDevice);
	
	id1_args->dimProjX=1000;
	id1_args->dimProjY=1000;
	id1_args->sparsity=100;
	id1_args->seedX=rand()%100;
	id1_args->seedY=rand()%100;
	// id1_args->len = model.keyX.size();

	hipMemcpy(d_args,id1_args,sizeof(struct Args_device),hipMemcpyHostToDevice);

	struct Rand_proj *out;
	hipMalloc((void **)&out,sizeof(struct Rand_proj));
	buildTree<<<1,1>>>(d_args,out);
	
	// struct Rand_proj *from_cuda=(struct Rand_proj*)malloc(sizeof(struct Rand_proj));

	// hipMemcpy(from_cuda,out,sizeof(struct Rand_proj),hipMemcpyDeviceToHost);
	
	// for()
	// cout<<*spar<<endl;

	// cout<<model.keyX[1000].size()<<endl;
	// cout<<getIndex(1,78,1000)<<endl;
	
	// cout<<getSign(1,78)<<endl;
	
    return 0;
}