#include "hip/hip_runtime.h"
#include<bits/stdc++.h>
#include<string.h>
// #include "RecordTextReader.cpp"
#include "LibsvmFileReader.cu"
#include<math.h>
#include <iostream>
#include <fstream>
using namespace std;

uint32_t MurmurHash2 ( const void * key, int len, uint32_t seed );
class CraftML
{
	public:
		
	vector<vector<float>> valuesX;
	vector<vector<int>> keyX;
	vector<vector<float>> valuesY;
	vector<vector<int>> keyY;
	
	void storeForLearning(unordered_map<int,float> x,unordered_map<int,float> y)
	{
		vector<float> valueX;
		vector<float> valueY;

		vector<int> indexX;
		vector<int> indexY;


		for (auto row : x) 
      	{
			indexX.push_back(row.first);
			valueX.push_back(row.second); 
		}

		for (auto row : y) 
      	{
			indexY.push_back(row.first);
			valueY.push_back(row.second); 
		}

		if(indexX.size()>0 && indexY.size()>0)
		{
			keyX.push_back(indexX);
			keyY.push_back(indexY);

			valuesX.push_back(valueX);
			valuesY.push_back(valueY);

		} 

	}
    void trainAlgoOnFileStandard(LibsvmFileReader &fileReader)
    {
        fileReader.openFile();
		
		// reinitialize();
		// SmallItem x;
		// SmallItem y;
		unordered_map<int,float> x;
		unordered_map<int,float> y;

		pair<unordered_map<int,float>,unordered_map<int,float>> x_y = fileReader.readNext();
		// boolean continueRead = fileReader.readNext();
		
		while(x_y.first.find(-1)==x_y.first.end())
		{
		// 	if(interrompre()) {
		// 		return;
		// 	}
			x = x_y.first;
			y = x_y.second;

			storeForLearning(x, y);
			x_y = fileReader.readNext();
		}
		cout<<keyX.size()<<endl;
		// for(int i=0;i<keyX.size();i++)
		// {
		// 	for(int j=0;j<keyX[i].size();j++)
		// 	{
		// 		cout<<keyX[i][j]<<" "<<valuesX[i][j]<<"; ";
		// 	}
		// 	cout<<endl;
		// }
		// indicationFinDePasse();
		// hasLearnt = true;
		// fileReader.closeFile();
    }
    

};

int get_length(int key)
{
	int len = 1;
	while(key>9)
	{
		len++;
		key/=10;
	}
	return len;
}

uint32_t MurmurHash2 ( const void * key, int len, uint32_t seed )
{
  /* 'm' and 'r' are mixing constants generated offline.
     They're not really 'magic', they just happen to work well.  */

  const uint32_t m = 0x5bd1e995;
  const int r = 24;

  /* Initialize the hash to a 'random' value */

  uint32_t h = seed ^ len;

  /* Mix 4 bytes at a time into the hash */

  const unsigned char * data = (const unsigned char *)key;

  while(len >= 4)
  {
    uint32_t k = *(uint32_t*)data;

    k *= m;
    k ^= k >> r;
    k *= m;

    h *= m;
    h ^= k;

    data += 4;
    len -= 4;
  }

  /* Handle the last few bytes of the input array  */

  switch(len)
  {
  case 3: h ^= data[2] << 16;
  case 2: h ^= data[1] << 8;
  case 1: h ^= data[0];
      h *= m;
  };

  /* Do a few final mixes of the hash to ensure the last few
  // bytes are well-incorporated.  */

  h ^= h >> 13;
  h *= m;
  h ^= h >> 15;

  return h;
} 

int getIndex(int key,int seed,int sizeMax){
	int len = get_length(key)+3;

	char hashkey[len];

	//key="azv"

	hashkey[0]='a';
	hashkey[1]='z';
	hashkey[2]='v';

	int i=1;
	while(key>0)
	{
		hashkey[len-i]=(char)(key%10+48);
		i++;
		key/=10;
	}

	return abs((int)MurmurHash2(hashkey,len,seed) % sizeMax);
}
float getSign(int key,int seed){
	int len = get_length(key)+3;

	char hashkey[len];

	//key="azv"

	hashkey[0]='a';
	hashkey[1]='z';
	hashkey[2]='v';

	int i=1;
	while(key>0)
	{
		hashkey[len-i]=(char)(key%10+48);
		i++;
		key/=10;
	}


	return (float)abs((int)MurmurHash2(hashkey, len, seed) % 2)*2 - 1;
}


int main(int argc,char** argv)
{
	string trainFile=argv[1];

    CraftML model ;

    LibsvmFileReader readerTrain;
					
	readerTrain.setFile(trainFile);

	// model.trainAlgoOnFileStandard(readerTrain);

	cout<<getIndex(1,78,1000)<<endl;
	
	cout<<getSign(1,78)<<endl;

    
    return 0;
}